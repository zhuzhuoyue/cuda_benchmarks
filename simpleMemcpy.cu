#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cinttypes>
#include <hip/hip_runtime.h>
#include "common.hh"


static __global__ void
f(const uint64_t a[], const uint64_t b[], uint64_t c[], int64_t N)
{
    int64_t index = threadIdx.x + blockIdx.x * blockDim.x;
    int64_t stride = blockDim.x * gridDim.x;

    for (int64_t i = index; i < N; i += stride) {
        c[i] = a[i] * b[i];
    }
}

static void
doit(const uint64_t a[], const uint64_t b[], uint64_t c[], int64_t N)
{
    int blockSize = 256;
    int64_t numBlocks = (N + blockSize - 1) / blockSize;

    f<<<numBlocks, blockSize>>>(a, b, c, N);
}

int
main(int argc, char *argv[])
{
    size_t N = 10000000;
    clock_t start_program, end_program;
    clock_t start, end;
    uint64_t *a, *b, *c;
    uint64_t *da, *db, *dc;
    size_t count;

    if (argc == 2) {
        N = checked_strtosize(argv[1]);
    }
    count = checked_mul(N, sizeof(uint64_t));

    /* Initialize context */
    check(hipHostMalloc(&a, 128));
    check(hipDeviceSynchronize());
    check(hipHostFree(a));

    start_program = clock();

    start = clock();
    check(hipHostMalloc(&a, count));
    check(hipHostMalloc(&b, count));
    check(hipHostMalloc(&c, count));
    end = clock();
    log("host: MallocHost", start, end);

    start = clock();
    for (size_t i = 0; i < N; i++) {
        a[i] = 3;
        b[i] = 5;
    }
    end = clock();
    log("host: init arrays", start, end);

    start = clock();
    check(hipMalloc(&da, count));
    check(hipMalloc(&db, count));
    check(hipMalloc(&dc, count));

    check(hipMemcpy(da, a, count, hipMemcpyHostToDevice));
    check(hipMemcpy(db, b, count, hipMemcpyHostToDevice));

    doit(da, db, dc, N);

    check(hipMemcpy(c, dc, count, hipMemcpyDeviceToHost));

    check(hipFree(da));
    check(hipFree(db));
    check(hipFree(dc));
    end = clock();
    log("device: malloc+copy+compute", start, end);

    start = clock();
    for (size_t i = 0; i < N; i++) {
        if (a[i] != 3 || b[i] != 5 || c[i] != 15) {
            fprintf(stderr, "unexpected result a: %lu  b: %lu  c: %lu\n",
                    a[i], b[i], c[i]);
            exit(1);
        }
    }
    end = clock();
    log("host: access all arrays", start, end);

    start = clock();
    for (size_t i = 0; i < N; i++) {
        if (a[i] != 3 || b[i] != 5 || c[i] != 15) {
            fprintf(stderr, "unexpected result a: %lu  b: %lu  c: %lu\n",
                    a[i], b[i], c[i]);
            exit(1);
        }
    }
    end = clock();
    log("host: access all arrays a second time", start, end);

    start = clock();
    check(hipHostFree(a));
    check(hipHostFree(b));
    check(hipHostFree(c));
    end = clock();
    log("host: free", start, end);

    end_program = clock();
    log("total", start_program, end_program);

    return 0;
}
